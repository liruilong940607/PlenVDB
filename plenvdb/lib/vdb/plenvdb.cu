#include "plenvdb.cuh"

void gpuAssert(hipError_t code, const char *file, int line, bool abort=false)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if( abort )
            exit(code);
    }
}

void gpuCheckKernelExecutionError( const char *file, int line)
{
    gpuAssert( hipPeekAtLastError(), file, line);
    gpuAssert( hipDeviceSynchronize(), file, line);    
}